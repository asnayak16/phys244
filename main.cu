#include "hip/hip_runtime.h"
//*****************************************************************************
//PHYS 244							main.cu	       				14th June 2017
//-----------------------------------------------------------------------------
//Author: Ganesh Ranganath Chandrasekar Iyer, Ashwin Nayak, Prashant Singh
//Objective: Implements the Marching Cube Algorithm using Nvidia's GP-GPU 
//Funtions: StatusLog, input_parameter_initialization, inputRead, 
//			marchingCubes,kernel,linearInterpolation, GetMemoryStatus
//          createOutputList,writeOutputList
//Compiler: nvcc
//Note: x64 Relese, compute_30,sm_30, /STACK:2000000, /HEAP:2000000
//Input: Name of the file which has the input parameters in command line
//		 iso_data.dat: File which has the volumetric data
//*****************************************************************************

/******************************************************************************
Disclaimer: The Tables are based on Cory Gene Bloyd formulation and algorithm
is based on Paul Bourke's article.

For additional details refer: http://paulbourke.net/geometry/polygonise/

Reference: Lorenson W.E. and Cline H.E., ‘Marching Cubes: A High-Resolution 3D 
		   Surface Construction Algorithm’, Computer Graphics (SIGGRAPH Proceed
		   ings), 1987, Vol 21-4, 163-169.
		   Suh, J.W., Kim, Y., ‘Accelerating MATLAB with GPU Computing’, 2013.
******************************************************************************/

//-----------------------------------------------------------------------------
// Including Header Files

#include"hip/hip_runtime.h"
#include""

#include <iostream>
#include <fstream>
#include <string>
#include <numeric>

#include <thrust/host_vector.h>
#include <thrust/fill.h>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include<thrust/sequence.h>
#include<thrust/execution_policy.h>
#include<thrust/transform.h>
#include"hip/hip_vector_types.h"
#include"vector_functions.h"
#include<thrust/device_ptr.h>
#include<thrust/system_error.h>

//-----------------------------------------------------------------------------
//Initialisation of Constants and Global Variables
//-----------------------------------------------------------------------------

//Maximum vertices 15
int MAX_VERTICES = 15;
//Initialising for use in GPU
__constant__ int MAX_VERTEX = 15;

/*
Output bin - contains unformatted vertex list
       counter - contains number of traingles for each voxel		
*/
float3* bin;
int* counter;

__constant__ unsigned int edgeTable[256] =
{
	0,  265,  515,  778, 1030, 1295, 1541, 1804,
	2060, 2309, 2575, 2822, 3082, 3331, 3593, 3840,
	400,  153,  915,  666, 1430, 1183, 1941, 1692,
	2460, 2197, 2975, 2710, 3482, 3219, 3993, 3728,
	560,  825,   51,  314, 1590, 1855, 1077, 1340,
	2620, 2869, 2111, 2358, 3642, 3891, 3129, 3376,
	928,  681,  419,  170, 1958, 1711, 1445, 1196,
	2988, 2725, 2479, 2214, 4010, 3747, 3497, 3232,
	1120, 1385, 1635, 1898,  102,  367,  613,  876,
	3180, 3429, 3695, 3942, 2154, 2403, 2665, 2912,
	1520, 1273, 2035, 1786,  502,  255, 1013,  764,
	3580, 3317, 4095, 3830, 2554, 2291, 3065, 2800,
	1616, 1881, 1107, 1370,  598,  863,   85,  348,
	3676, 3925, 3167, 3414, 2650, 2899, 2137, 2384,
	1984, 1737, 1475, 1226,  966,  719,  453,  204,
	4044, 3781, 3535, 3270, 3018, 2755, 2505, 2240,
	2240, 2505, 2755, 3018, 3270, 3535, 3781, 4044,
	204,  453,  719,  966, 1226, 1475, 1737, 1984,
	2384, 2137, 2899, 2650, 3414, 3167, 3925, 3676,
	348,   85,  863,  598, 1370, 1107, 1881, 1616,
	2800, 3065, 2291, 2554, 3830, 4095, 3317, 3580,
	764, 1013,  255,  502, 1786, 2035, 1273, 1520,
	2912, 2665, 2403, 2154, 3942, 3695, 3429, 3180,
	876,  613,  367,  102, 1898, 1635, 1385, 1120,
	3232, 3497, 3747, 4010, 2214, 2479, 2725, 2988,
	1196, 1445, 1711, 1958,  170,  419,  681,  928,
	3376, 3129, 3891, 3642, 2358, 2111, 2869, 2620,
	1340, 1077, 1855, 1590,  314,   51,  825,  560,
	3728, 3993, 3219, 3482, 2710, 2975, 2197, 2460,
	1692, 1941, 1183, 1430,  666,  915,  153,  400,
	3840, 3593, 3331, 3082, 2822, 2575, 2309, 2060,
	1804, 1541, 1295, 1030,  778,  515,  265,    0
};

__constant__ int triTable[256][16] =
{
	{ -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 0,  8,  3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 0,  1,  9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 1,  8,  3,  9,  8,  1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 1,  2, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 0,  8,  3,  1,  2, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 9,  2, 10,  0,  2,  9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 2,  8,  3,  2, 10,  8, 10,  9,  8, -1, -1, -1, -1, -1, -1, -1 },
	{ 3, 11,  2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 0, 11,  2,  8, 11,  0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 1,  9,  0,  2,  3, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 1, 11,  2,  1,  9, 11,  9,  8, 11, -1, -1, -1, -1, -1, -1, -1 },
	{ 3, 10,  1, 11, 10,  3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 0, 10,  1,  0,  8, 10,  8, 11, 10, -1, -1, -1, -1, -1, -1, -1 },
	{ 3,  9,  0,  3, 11,  9, 11, 10,  9, -1, -1, -1, -1, -1, -1, -1 },
	{ 9,  8, 10, 10,  8, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 4,  7,  8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 4,  3,  0,  7,  3,  4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 0,  1,  9,  8,  4,  7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 4,  1,  9,  4,  7,  1,  7,  3,  1, -1, -1, -1, -1, -1, -1, -1 },
	{ 1,  2, 10,  8,  4,  7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 3,  4,  7,  3,  0,  4,  1,  2, 10, -1, -1, -1, -1, -1, -1, -1 },
	{ 9,  2, 10,  9,  0,  2,  8,  4,  7, -1, -1, -1, -1, -1, -1, -1 },
	{ 2, 10,  9,  2,  9,  7,  2,  7,  3,  7,  9,  4, -1, -1, -1, -1 },
	{ 8,  4,  7,  3, 11,  2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 11,  4,  7, 11,  2,  4,  2,  0,  4, -1, -1, -1, -1, -1, -1, -1 },
	{ 9,  0,  1,  8,  4,  7,  2,  3, 11, -1, -1, -1, -1, -1, -1, -1 },
	{ 4,  7, 11,  9,  4, 11,  9, 11,  2,  9,  2,  1, -1, -1, -1, -1 },
	{ 3, 10,  1,  3, 11, 10,  7,  8,  4, -1, -1, -1, -1, -1, -1, -1 },
	{ 1, 11, 10,  1,  4, 11,  1,  0,  4,  7, 11,  4, -1, -1, -1, -1 },
	{ 4,  7,  8,  9,  0, 11,  9, 11, 10, 11,  0,  3, -1, -1, -1, -1 },
	{ 4,  7, 11,  4, 11,  9,  9, 11, 10, -1, -1, -1, -1, -1, -1, -1 },
	{ 9,  5,  4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 9,  5,  4,  0,  8,  3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 0,  5,  4,  1,  5,  0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 8,  5,  4,  8,  3,  5,  3,  1,  5, -1, -1, -1, -1, -1, -1, -1 },
	{ 1,  2, 10,  9,  5,  4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 3,  0,  8,  1,  2, 10,  4,  9,  5, -1, -1, -1, -1, -1, -1, -1 },
	{ 5,  2, 10,  5,  4,  2,  4,  0,  2, -1, -1, -1, -1, -1, -1, -1 },
	{ 2, 10,  5,  3,  2,  5,  3,  5,  4,  3,  4,  8, -1, -1, -1, -1 },
	{ 9,  5,  4,  2,  3, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 0, 11,  2,  0,  8, 11,  4,  9,  5, -1, -1, -1, -1, -1, -1, -1 },
	{ 0,  5,  4,  0,  1,  5,  2,  3, 11, -1, -1, -1, -1, -1, -1, -1 },
	{ 2,  1,  5,  2,  5,  8,  2,  8, 11,  4,  8,  5, -1, -1, -1, -1 },
	{ 10,  3, 11, 10,  1,  3,  9,  5,  4, -1, -1, -1, -1, -1, -1, -1 },
	{ 4,  9,  5,  0,  8,  1,  8, 10,  1,  8, 11, 10, -1, -1, -1, -1 },
	{ 5,  4,  0,  5,  0, 11,  5, 11, 10, 11,  0,  3, -1, -1, -1, -1 },
	{ 5,  4,  8,  5,  8, 10, 10,  8, 11, -1, -1, -1, -1, -1, -1, -1 },
	{ 9,  7,  8,  5,  7,  9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 9,  3,  0,  9,  5,  3,  5,  7,  3, -1, -1, -1, -1, -1, -1, -1 },
	{ 0,  7,  8,  0,  1,  7,  1,  5,  7, -1, -1, -1, -1, -1, -1, -1 },
	{ 1,  5,  3,  3,  5,  7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 9,  7,  8,  9,  5,  7, 10,  1,  2, -1, -1, -1, -1, -1, -1, -1 },
	{ 10,  1,  2,  9,  5,  0,  5,  3,  0,  5,  7,  3, -1, -1, -1, -1 },
	{ 8,  0,  2,  8,  2,  5,  8,  5,  7, 10,  5,  2, -1, -1, -1, -1 },
	{ 2, 10,  5,  2,  5,  3,  3,  5,  7, -1, -1, -1, -1, -1, -1, -1 },
	{ 7,  9,  5,  7,  8,  9,  3, 11,  2, -1, -1, -1, -1, -1, -1, -1 },
	{ 9,  5,  7,  9,  7,  2,  9,  2,  0,  2,  7, 11, -1, -1, -1, -1 },
	{ 2,  3, 11,  0,  1,  8,  1,  7,  8,  1,  5,  7, -1, -1, -1, -1 },
	{ 11,  2,  1, 11,  1,  7,  7,  1,  5, -1, -1, -1, -1, -1, -1, -1 },
	{ 9,  5,  8,  8,  5,  7, 10,  1,  3, 10,  3, 11, -1, -1, -1, -1 },
	{ 5,  7,  0,  5,  0,  9,  7, 11,  0,  1,  0, 10, 11, 10,  0, -1 },
	{ 11, 10,  0, 11,  0,  3, 10,  5,  0,  8,  0,  7,  5,  7,  0, -1 },
	{ 11, 10,  5,  7, 11,  5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 10,  6,  5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 0,  8,  3,  5, 10,  6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 9,  0,  1,  5, 10,  6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 1,  8,  3,  1,  9,  8,  5, 10,  6, -1, -1, -1, -1, -1, -1, -1 },
	{ 1,  6,  5,  2,  6,  1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 1,  6,  5,  1,  2,  6,  3,  0,  8, -1, -1, -1, -1, -1, -1, -1 },
	{ 9,  6,  5,  9,  0,  6,  0,  2,  6, -1, -1, -1, -1, -1, -1, -1 },
	{ 5,  9,  8,  5,  8,  2,  5,  2,  6,  3,  2,  8, -1, -1, -1, -1 },
	{ 2,  3, 11, 10,  6,  5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 11,  0,  8, 11,  2,  0, 10,  6,  5, -1, -1, -1, -1, -1, -1, -1 },
	{ 0,  1,  9,  2,  3, 11,  5, 10,  6, -1, -1, -1, -1, -1, -1, -1 },
	{ 5, 10,  6,  1,  9,  2,  9, 11,  2,  9,  8, 11, -1, -1, -1, -1 },
	{ 6,  3, 11,  6,  5,  3,  5,  1,  3, -1, -1, -1, -1, -1, -1, -1 },
	{ 0,  8, 11,  0, 11,  5,  0,  5,  1,  5, 11,  6, -1, -1, -1, -1 },
	{ 3, 11,  6,  0,  3,  6,  0,  6,  5,  0,  5,  9, -1, -1, -1, -1 },
	{ 6,  5,  9,  6,  9, 11, 11,  9,  8, -1, -1, -1, -1, -1, -1, -1 },
	{ 5, 10,  6,  4,  7,  8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 4,  3,  0,  4,  7,  3,  6,  5, 10, -1, -1, -1, -1, -1, -1, -1 },
	{ 1,  9,  0,  5, 10,  6,  8,  4,  7, -1, -1, -1, -1, -1, -1, -1 },
	{ 10,  6,  5,  1,  9,  7,  1,  7,  3,  7,  9,  4, -1, -1, -1, -1 },
	{ 6,  1,  2,  6,  5,  1,  4,  7,  8, -1, -1, -1, -1, -1, -1, -1 },
	{ 1,  2,  5,  5,  2,  6,  3,  0,  4,  3,  4,  7, -1, -1, -1, -1 },
	{ 8,  4,  7,  9,  0,  5,  0,  6,  5,  0,  2,  6, -1, -1, -1, -1 },
	{ 7,  3,  9,  7,  9,  4,  3,  2,  9,  5,  9,  6,  2,  6,  9, -1 },
	{ 3, 11,  2,  7,  8,  4, 10,  6,  5, -1, -1, -1, -1, -1, -1, -1 },
	{ 5, 10,  6,  4,  7,  2,  4,  2,  0,  2,  7, 11, -1, -1, -1, -1 },
	{ 0,  1,  9,  4,  7,  8,  2,  3, 11,  5, 10,  6, -1, -1, -1, -1 },
	{ 9,  2,  1,  9, 11,  2,  9,  4, 11,  7, 11,  4,  5, 10,  6, -1 },
	{ 8,  4,  7,  3, 11,  5,  3,  5,  1,  5, 11,  6, -1, -1, -1, -1 },
	{ 5,  1, 11,  5, 11,  6,  1,  0, 11,  7, 11,  4,  0,  4, 11, -1 },
	{ 0,  5,  9,  0,  6,  5,  0,  3,  6, 11,  6,  3,  8,  4,  7, -1 },
	{ 6,  5,  9,  6,  9, 11,  4,  7,  9,  7, 11,  9, -1, -1, -1, -1 },
	{ 10,  4,  9,  6,  4, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 4, 10,  6,  4,  9, 10,  0,  8,  3, -1, -1, -1, -1, -1, -1, -1 },
	{ 10,  0,  1, 10,  6,  0,  6,  4,  0, -1, -1, -1, -1, -1, -1, -1 },
	{ 8,  3,  1,  8,  1,  6,  8,  6,  4,  6,  1, 10, -1, -1, -1, -1 },
	{ 1,  4,  9,  1,  2,  4,  2,  6,  4, -1, -1, -1, -1, -1, -1, -1 },
	{ 3,  0,  8,  1,  2,  9,  2,  4,  9,  2,  6,  4, -1, -1, -1, -1 },
	{ 0,  2,  4,  4,  2,  6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 8,  3,  2,  8,  2,  4,  4,  2,  6, -1, -1, -1, -1, -1, -1, -1 },
	{ 10,  4,  9, 10,  6,  4, 11,  2,  3, -1, -1, -1, -1, -1, -1, -1 },
	{ 0,  8,  2,  2,  8, 11,  4,  9, 10,  4, 10,  6, -1, -1, -1, -1 },
	{ 3, 11,  2,  0,  1,  6,  0,  6,  4,  6,  1, 10, -1, -1, -1, -1 },
	{ 6,  4,  1,  6,  1, 10,  4,  8,  1,  2,  1, 11,  8, 11,  1, -1 },
	{ 9,  6,  4,  9,  3,  6,  9,  1,  3, 11,  6,  3, -1, -1, -1, -1 },
	{ 8, 11,  1,  8,  1,  0, 11,  6,  1,  9,  1,  4,  6,  4,  1, -1 },
	{ 3, 11,  6,  3,  6,  0,  0,  6,  4, -1, -1, -1, -1, -1, -1, -1 },
	{ 6,  4,  8, 11,  6,  8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 7, 10,  6,  7,  8, 10,  8,  9, 10, -1, -1, -1, -1, -1, -1, -1 },
	{ 0,  7,  3,  0, 10,  7,  0,  9, 10,  6,  7, 10, -1, -1, -1, -1 },
	{ 10,  6,  7,  1, 10,  7,  1,  7,  8,  1,  8,  0, -1, -1, -1, -1 },
	{ 10,  6,  7, 10,  7,  1,  1,  7,  3, -1, -1, -1, -1, -1, -1, -1 },
	{ 1,  2,  6,  1,  6,  8,  1,  8,  9,  8,  6,  7, -1, -1, -1, -1 },
	{ 2,  6,  9,  2,  9,  1,  6,  7,  9,  0,  9,  3,  7,  3,  9, -1 },
	{ 7,  8,  0,  7,  0,  6,  6,  0,  2, -1, -1, -1, -1, -1, -1, -1 },
	{ 7,  3,  2,  6,  7,  2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 2,  3, 11, 10,  6,  8, 10,  8,  9,  8,  6,  7, -1, -1, -1, -1 },
	{ 2,  0,  7,  2,  7, 11,  0,  9,  7,  6,  7, 10,  9, 10,  7, -1 },
	{ 1,  8,  0,  1,  7,  8,  1, 10,  7,  6,  7, 10,  2,  3, 11, -1 },
	{ 11,  2,  1, 11,  1,  7, 10,  6,  1,  6,  7,  1, -1, -1, -1, -1 },
	{ 8,  9,  6,  8,  6,  7,  9,  1,  6, 11,  6,  3,  1,  3,  6, -1 },
	{ 0,  9,  1, 11,  6,  7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 7,  8,  0,  7,  0,  6,  3, 11,  0, 11,  6,  0, -1, -1, -1, -1 },
	{ 7, 11,  6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 7,  6, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 3,  0,  8, 11,  7,  6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 0,  1,  9, 11,  7,  6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 8,  1,  9,  8,  3,  1, 11,  7,  6, -1, -1, -1, -1, -1, -1, -1 },
	{ 10,  1,  2,  6, 11,  7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 1,  2, 10,  3,  0,  8,  6, 11,  7, -1, -1, -1, -1, -1, -1, -1 },
	{ 2,  9,  0,  2, 10,  9,  6, 11,  7, -1, -1, -1, -1, -1, -1, -1 },
	{ 6, 11,  7,  2, 10,  3, 10,  8,  3, 10,  9,  8, -1, -1, -1, -1 },
	{ 7,  2,  3,  6,  2,  7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 7,  0,  8,  7,  6,  0,  6,  2,  0, -1, -1, -1, -1, -1, -1, -1 },
	{ 2,  7,  6,  2,  3,  7,  0,  1,  9, -1, -1, -1, -1, -1, -1, -1 },
	{ 1,  6,  2,  1,  8,  6,  1,  9,  8,  8,  7,  6, -1, -1, -1, -1 },
	{ 10,  7,  6, 10,  1,  7,  1,  3,  7, -1, -1, -1, -1, -1, -1, -1 },
	{ 10,  7,  6,  1,  7, 10,  1,  8,  7,  1,  0,  8, -1, -1, -1, -1 },
	{ 0,  3,  7,  0,  7, 10,  0, 10,  9,  6, 10,  7, -1, -1, -1, -1 },
	{ 7,  6, 10,  7, 10,  8,  8, 10,  9, -1, -1, -1, -1, -1, -1, -1 },
	{ 6,  8,  4, 11,  8,  6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 3,  6, 11,  3,  0,  6,  0,  4,  6, -1, -1, -1, -1, -1, -1, -1 },
	{ 8,  6, 11,  8,  4,  6,  9,  0,  1, -1, -1, -1, -1, -1, -1, -1 },
	{ 9,  4,  6,  9,  6,  3,  9,  3,  1, 11,  3,  6, -1, -1, -1, -1 },
	{ 6,  8,  4,  6, 11,  8,  2, 10,  1, -1, -1, -1, -1, -1, -1, -1 },
	{ 1,  2, 10,  3,  0, 11,  0,  6, 11,  0,  4,  6, -1, -1, -1, -1 },
	{ 4, 11,  8,  4,  6, 11,  0,  2,  9,  2, 10,  9, -1, -1, -1, -1 },
	{ 10,  9,  3, 10,  3,  2,  9,  4,  3, 11,  3,  6,  4,  6,  3, -1 },
	{ 8,  2,  3,  8,  4,  2,  4,  6,  2, -1, -1, -1, -1, -1, -1, -1 },
	{ 0,  4,  2,  4,  6,  2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 1,  9,  0,  2,  3,  4,  2,  4,  6,  4,  3,  8, -1, -1, -1, -1 },
	{ 1,  9,  4,  1,  4,  2,  2,  4,  6, -1, -1, -1, -1, -1, -1, -1 },
	{ 8,  1,  3,  8,  6,  1,  8,  4,  6,  6, 10,  1, -1, -1, -1, -1 },
	{ 10,  1,  0, 10,  0,  6,  6,  0,  4, -1, -1, -1, -1, -1, -1, -1 },
	{ 4,  6,  3,  4,  3,  8,  6, 10,  3,  0,  3,  9, 10,  9,  3, -1 },
	{ 10,  9,  4,  6, 10,  4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 4,  9,  5,  7,  6, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 0,  8,  3,  4,  9,  5, 11,  7,  6, -1, -1, -1, -1, -1, -1, -1 },
	{ 5,  0,  1,  5,  4,  0,  7,  6, 11, -1, -1, -1, -1, -1, -1, -1 },
	{ 11,  7,  6,  8,  3,  4,  3,  5,  4,  3,  1,  5, -1, -1, -1, -1 },
	{ 9,  5,  4, 10,  1,  2,  7,  6, 11, -1, -1, -1, -1, -1, -1, -1 },
	{ 6, 11,  7,  1,  2, 10,  0,  8,  3,  4,  9,  5, -1, -1, -1, -1 },
	{ 7,  6, 11,  5,  4, 10,  4,  2, 10,  4,  0,  2, -1, -1, -1, -1 },
	{ 3,  4,  8,  3,  5,  4,  3,  2,  5, 10,  5,  2, 11,  7,  6, -1 },
	{ 7,  2,  3,  7,  6,  2,  5,  4,  9, -1, -1, -1, -1, -1, -1, -1 },
	{ 9,  5,  4,  0,  8,  6,  0,  6,  2,  6,  8,  7, -1, -1, -1, -1 },
	{ 3,  6,  2,  3,  7,  6,  1,  5,  0,  5,  4,  0, -1, -1, -1, -1 },
	{ 6,  2,  8,  6,  8,  7,  2,  1,  8,  4,  8,  5,  1,  5,  8, -1 },
	{ 9,  5,  4, 10,  1,  6,  1,  7,  6,  1,  3,  7, -1, -1, -1, -1 },
	{ 1,  6, 10,  1,  7,  6,  1,  0,  7,  8,  7,  0,  9,  5,  4, -1 },
	{ 4,  0, 10,  4, 10,  5,  0,  3, 10,  6, 10,  7,  3,  7, 10, -1 },
	{ 7,  6, 10,  7, 10,  8,  5,  4, 10,  4,  8, 10, -1, -1, -1, -1 },
	{ 6,  9,  5,  6, 11,  9, 11,  8,  9, -1, -1, -1, -1, -1, -1, -1 },
	{ 3,  6, 11,  0,  6,  3,  0,  5,  6,  0,  9,  5, -1, -1, -1, -1 },
	{ 0, 11,  8,  0,  5, 11,  0,  1,  5,  5,  6, 11, -1, -1, -1, -1 },
	{ 6, 11,  3,  6,  3,  5,  5,  3,  1, -1, -1, -1, -1, -1, -1, -1 },
	{ 1,  2, 10,  9,  5, 11,  9, 11,  8, 11,  5,  6, -1, -1, -1, -1 },
	{ 0, 11,  3,  0,  6, 11,  0,  9,  6,  5,  6,  9,  1,  2, 10, -1 },
	{ 11,  8,  5, 11,  5,  6,  8,  0,  5, 10,  5,  2,  0,  2,  5, -1 },
	{ 6, 11,  3,  6,  3,  5,  2, 10,  3, 10,  5,  3, -1, -1, -1, -1 },
	{ 5,  8,  9,  5,  2,  8,  5,  6,  2,  3,  8,  2, -1, -1, -1, -1 },
	{ 9,  5,  6,  9,  6,  0,  0,  6,  2, -1, -1, -1, -1, -1, -1, -1 },
	{ 1,  5,  8,  1,  8,  0,  5,  6,  8,  3,  8,  2,  6,  2,  8, -1 },
	{ 1,  5,  6,  2,  1,  6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 1,  3,  6,  1,  6, 10,  3,  8,  6,  5,  6,  9,  8,  9,  6, -1 },
	{ 10,  1,  0, 10,  0,  6,  9,  5,  0,  5,  6,  0, -1, -1, -1, -1 },
	{ 0,  3,  8,  5,  6, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 10,  5,  6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 11,  5, 10,  7,  5, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 11,  5, 10, 11,  7,  5,  8,  3,  0, -1, -1, -1, -1, -1, -1, -1 },
	{ 5, 11,  7,  5, 10, 11,  1,  9,  0, -1, -1, -1, -1, -1, -1, -1 },
	{ 10,  7,  5, 10, 11,  7,  9,  8,  1,  8,  3,  1, -1, -1, -1, -1 },
	{ 11,  1,  2, 11,  7,  1,  7,  5,  1, -1, -1, -1, -1, -1, -1, -1 },
	{ 0,  8,  3,  1,  2,  7,  1,  7,  5,  7,  2, 11, -1, -1, -1, -1 },
	{ 9,  7,  5,  9,  2,  7,  9,  0,  2,  2, 11,  7, -1, -1, -1, -1 },
	{ 7,  5,  2,  7,  2, 11,  5,  9,  2,  3,  2,  8,  9,  8,  2, -1 },
	{ 2,  5, 10,  2,  3,  5,  3,  7,  5, -1, -1, -1, -1, -1, -1, -1 },
	{ 8,  2,  0,  8,  5,  2,  8,  7,  5, 10,  2,  5, -1, -1, -1, -1 },
	{ 9,  0,  1,  5, 10,  3,  5,  3,  7,  3, 10,  2, -1, -1, -1, -1 },
	{ 9,  8,  2,  9,  2,  1,  8,  7,  2, 10,  2,  5,  7,  5,  2, -1 },
	{ 1,  3,  5,  3,  7,  5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 0,  8,  7,  0,  7,  1,  1,  7,  5, -1, -1, -1, -1, -1, -1, -1 },
	{ 9,  0,  3,  9,  3,  5,  5,  3,  7, -1, -1, -1, -1, -1, -1, -1 },
	{ 9,  8,  7,  5,  9,  7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 5,  8,  4,  5, 10,  8, 10, 11,  8, -1, -1, -1, -1, -1, -1, -1 },
	{ 5,  0,  4,  5, 11,  0,  5, 10, 11, 11,  3,  0, -1, -1, -1, -1 },
	{ 0,  1,  9,  8,  4, 10,  8, 10, 11, 10,  4,  5, -1, -1, -1, -1 },
	{ 10, 11,  4, 10,  4,  5, 11,  3,  4,  9,  4,  1,  3,  1,  4, -1 },
	{ 2,  5,  1,  2,  8,  5,  2, 11,  8,  4,  5,  8, -1, -1, -1, -1 },
	{ 0,  4, 11,  0, 11,  3,  4,  5, 11,  2, 11,  1,  5 , 1, 11, -1 },
	{ 0,  2,  5,  0,  5,  9,  2, 11,  5,  4,  5,  8, 11,  8,  5, -1 },
	{ 9,  4,  5,  2, 11,  3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 2,  5, 10,  3,  5,  2,  3,  4,  5,  3,  8,  4, -1, -1, -1, -1 },
	{ 5, 10,  2,  5,  2,  4,  4,  2,  0, -1, -1, -1, -1, -1, -1, -1 },
	{ 3, 10,  2,  3,  5, 10,  3,  8,  5,  4,  5,  8,  0,  1,  9, -1 },
	{ 5, 10,  2,  5,  2,  4,  1,  9,  2,  9,  4,  2, -1, -1, -1, -1 },
	{ 8,  4,  5,  8,  5,  3,  3,  5,  1, -1, -1, -1, -1, -1, -1, -1 },
	{ 0,  4,  5,  1,  0,  5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 8,  4,  5,  8,  5,  3,  9,  0,  5,  0,  3,  5, -1, -1, -1, -1 },
	{ 9,  4,  5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 4, 11,  7,  4,  9, 11,  9, 10, 11, -1, -1, -1, -1, -1, -1, -1 },
	{ 0,  8,  3,  4,  9,  7,  9, 11,  7,  9, 10, 11, -1, -1, -1, -1 },
	{ 1, 10, 11,  1, 11,  4,  1,  4,  0,  7,  4, 11, -1, -1, -1, -1 },
	{ 3,  1,  4,  3,  4,  8,  1, 10,  4,  7,  4, 11, 10, 11,  4, -1 },
	{ 4, 11,  7,  9, 11,  4,  9,  2, 11,  9,  1,  2, -1, -1, -1, -1 },
	{ 9,  7,  4,  9, 11,  7,  9,  1, 11,  2, 11,  1,  0,  8,  3, -1 },
	{ 11,  7,  4, 11,  4,  2,  2,  4,  0, -1, -1, -1, -1, -1, -1, -1 },
	{ 11,  7,  4, 11,  4,  2,  8,  3,  4,  3,  2,  4, -1, -1, -1, -1 },
	{ 2,  9, 10,  2,  7,  9,  2,  3,  7,  7,  4,  9, -1, -1, -1, -1 },
	{ 9, 10,  7,  9,  7,  4, 10,  2,  7,  8,  7,  0,  2,  0,  7, -1 },
	{ 3,  7, 10,  3, 10,  2,  7,  4, 10,  1, 10,  0,  4,  0, 10, -1 },
	{ 1, 10,  2,  8,  7,  4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 4,  9,  1,  4,  1,  7,  7,  1,  3, -1, -1, -1, -1, -1, -1, -1 },
	{ 4,  9,  1,  4,  1,  7,  0,  8,  1,  8,  7,  1, -1, -1, -1, -1 },
	{ 4,  0,  3,  7,  4,  3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 4,  8,  7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 9, 10,  8, 10, 11,  8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 3,  0,  9,  3,  9, 11, 11,  9, 10, -1, -1, -1, -1, -1, -1, -1 },
	{ 0,  1, 10,  0, 10,  8,  8, 10, 11, -1, -1, -1, -1, -1, -1, -1 },
	{ 3,  1, 10, 11,  3, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 1,  2, 11,  1, 11,  9,  9, 11,  8, -1, -1, -1, -1, -1, -1, -1 },
	{ 3,  0,  9,  3,  9, 11,  1,  2,  9,  2, 11,  9, -1, -1, -1, -1 },
	{ 0,  2, 11,  8,  0, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 3,  2, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 2,  3,  8,  2,  8, 10, 10,  8,  9, -1, -1, -1, -1, -1, -1, -1 },
	{ 9, 10,  2,  0,  9,  2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 2,  3,  8,  2,  8, 10,  0,  1,  8,  1, 10,  8, -1, -1, -1, -1 },
	{ 1, 10,  2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 1,  3,  8,  9,  1,  8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 0,  9,  1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ 0,  3,  8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
	{ -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 }
};
//-----------------------------------------------------------------------------
/*****************************************************************************/
//-----------------------------------------------------------------------------
//Declaring Functions and Structures
//-----------------------------------------------------------------------------
/*
* Name: inputParameters
* Type: struct
* Data Members: x_dim,y_dim,z_dim - Size of volumetric dataset along X,Y,and Z
*				var_start, dvar - Starting position, Delta
				                  Where var is in {x,y,z}
*               isovalue - Value for which iso-surface must be rendered
*               InputFile: Name of the file containing the volumetric dataset
*				OutputVertexFile: Name of the output file that contains
								  co-ordinates of vertices
				OutputIndexFile: Name of the output file that contains
								 vertex connectivity map
* Description: The structure <inputParameters> stores the dimension of input
*			   datset, and name of the input / output files.
*/
struct inputParameters {
	unsigned int x_dim, y_dim, z_dim;
	float x_start, y_start, z_start;
	float dx, dy, dz;
	float isovalue;
	std::string InputFile, OutputVertexFile, OutputIndexFile;
};
//-----------------------------------------------------------------------------
/*
* Function Name: StatusLog
* Return: void
* Arguments: const std::string - Contains the status message
*
* Description: The function accepts the message passed to it as a string and
*              prints it to the log file
*/
void StatusLog(const std::string);
//-----------------------------------------------------------------------------
/*
* Function Name: input_parameter_initialization
* Return: inputParameters
* Argument: const std::string - Contains the name of the file which has the
			input parameters
* Description: The functions initialises the input parameters
*/
inputParameters input_parameter_initialization(const std::string);
//-----------------------------------------------------------------------------
/*
* Function Name: inputRead
* Return: void
* Arguments: inputParameters: A structure containing the name of input
			 parameters
			 thrust::host_vector<float4> &: Host Vector 
* Description: The function reads the input data file and initialises a float4
			   vector (x,y,z,data) of size xdim * ydim * zdim

			   Row Major Order: [i][j][k] = i * ydim * zdim + j * zdim + k
*/
void inputRead(inputParameters, thrust::host_vector<float4> &);
//-----------------------------------------------------------------------------
/*
* Function Name: marchingCubes
* Return: void
* Arguments: inputParameters: A structure containing input parameters
             thrust::host_vector<float4>&: host_vector - [x,y,z,gridValue]
* Description: Function launches the kernel
*/
void marchingCubes(thrust::host_vector<float4>&, inputParameters);
//-----------------------------------------------------------------------------
/*
* Function Name: kernel
* Return: void
* Arguments: float - isovalue for the surface,
			 unsigned int - x,y,z dimension,
			 float4* - Device Vector (x,y,z,gridValue),
			 float3* - Vertex Bins,
			 int* - Traingle Counter
* Description: Performs Marching Cubes
*/
__global__ void kernel(float, unsigned int, unsigned int, unsigned int,
	float4*, float3*, int*);
//-----------------------------------------------------------------------------
/*
* Function Name: linearInterpolation
* Return: float3
* Arguments: float - Isovalue
			 float4 - Data of Voxel A
			 float4 - Data of Voxel B
* Description: Performs linear interpolation and finds position
*/
__device__ float3 linearInterpolation(float, float4, float4);
//-----------------------------------------------------------------------------
/*
* Function Name: writeOutputList
* Return: void
* Arguments: const int - totalSize  = xdim * ydim * zdim,
			 std::string - Vextex Output list filename,
			 std::string - Index Output list filename,
* Description: Prints the output lists to respective files
*/
void writeOutputList(const int, std::string, std::string);
//-----------------------------------------------------------------------------
/*
* Function Name: GetMemoryStatus
* Return: void
* Arguments: -
* Description: Prints the current available, used and total memory of the GPU
*/
void GetMemoryStatus();
//-----------------------------------------------------------------------------
/*****************************************************************************/
//-----------------------------------------------------------------------------
//Function Definitions
void StatusLog(const std::string status)
{
	//std::cout << status << std::endl;

	//Opening a file for output and seek to end before write
	std::ofstream LogFile("log.txt", std::ios::out | std::ios::app);

	//Checking if the file is open
	if (LogFile.is_open())
	{
		//Printing the status message to the file
		LogFile << status << std::endl;
	}
	else {
		//Printing error message to the screen when file fails to open
		std::cerr << "Error: Failed to open log file" << std::endl;
		exit(-1);
	}

	//Closing the output file stream
	LogFile.close();
}

inputParameters input_parameter_initialization(const std::string FileName)
{
	//Declaring an object of type <inputParameters>
	inputParameters data;

	//Opening a file for input
	std::ifstream ParamaterFile(FileName.c_str(), std::ios::in);

	//Checking if the file is open
	if (ParamaterFile.is_open())
	{
		
		//Reading the data from input file <FileName> and initialsing the
		//data members of the structure

		ParamaterFile >> data.x_dim >> data.y_dim >> data.z_dim
			>> data.x_start >> data.y_start >> data.z_start
			>> data.dx >> data.dy >> data.dz
			>> data.isovalue >> data.InputFile
			>> data.OutputVertexFile >> data.OutputIndexFile;

		//Updating status
		StatusLog("Initiliased the input parameters");
	}
	else
	{
		//Updating error
		StatusLog("Failed to open " + FileName);
		exit(-1);
	}

	//Closing the input file stream
	ParamaterFile.close();

	//Returning the structure
	return data;
}


void inputRead(inputParameters data, thrust::host_vector<float4> &host_vec)
{
	//Allocating Memory to X,Y,Z   
	thrust::host_vector<float> X(data.x_dim);
	thrust::host_vector<float> Y(data.y_dim);
	thrust::host_vector<float> Z(data.z_dim);


	//Generating X,Y,Z sequence
	thrust::sequence(thrust::host, 
					X.begin(), X.begin() + data.x_dim, data.x_start, data.dx);
	thrust::sequence(thrust::host,
					Y.begin(), Y.begin() + data.y_dim, data.y_start, data.dy);
	thrust::sequence(thrust::host,
					Z.begin(), Z.begin() + data.z_dim, data.z_start, data.dz);

	StatusLog("Generated Sequence for X,Y,Z");

	//Opening a file for reading the data
	std::ifstream inputFile(data.InputFile.c_str(), std::ios::in);

	//Checking if the file is open
	if (inputFile.is_open())
	{
		//Updating Status 
		StatusLog("Opened the file " + data.InputFile);

		//Temporary variables
		float temp;

		for (unsigned int x_index = 0; x_index < data.x_dim; x_index++) {
			for (unsigned int y_index = 0; y_index < data.y_dim; y_index++) {
				for (unsigned int z_index=0; z_index < data.z_dim; z_index++) {
					//Checking if data is read
					if (inputFile >> temp) {
						//Initialising host_vec
						host_vec.push_back(make_float4(
							X[x_index], Y[y_index], Z[z_index], temp));
					}
					else {
						StatusLog("Error: Failed to read " + data.InputFile);
						goto EXIT;
					}

				}
			}
		}

		StatusLog("Input initialised");
	}
	else {
		StatusLog("Failed to open:" + data.InputFile);
		exit(-1);
	}

EXIT:
	//Closing Stream
	inputFile.close();
	//Deleting memory allocated to X,Y,Z
	try {
		X.clear();
		X.shrink_to_fit();
		Y.clear();
		Y.shrink_to_fit();
		Z.clear();
		Z.shrink_to_fit();
	}
	catch (thrust::system_error e) {
		StatusLog(e.what());
		exit(-1);
	}
	return;
}

void GetMemoryStatus() {
	size_t available;
	size_t total;
	size_t used;

	hipMemGetInfo(&available, &total);
	used = total - available;

	//Updating Memory Usage
	StatusLog("Used Memory: " + std::to_string(used));
	StatusLog("Available Memory: " + std::to_string(available));
	StatusLog("Total Memory: " + std::to_string(total));
	StatusLog("****************************************");

	return;
}

__device__ float3 linearInterpolation(float isovalue, float4 voxelA,
	float4 voxelB) {
	const float scale = (isovalue - voxelA.w) / (voxelB.w - voxelA.w);

	//Coordinates of position will be in float3
	float3 position;

	//Initialising position
	position.x = voxelA.x + scale * (voxelB.x - voxelA.x);
	position.y = voxelA.y + scale * (voxelB.y - voxelA.y);
	position.z = voxelA.z + scale * (voxelB.z - voxelA.z);

	return position;
}


__global__ void kernel(float isovalue, unsigned int xdim, unsigned int ydim,
	unsigned int zdim, float4* device, float3* vertex, int* triangle)
{

	//id has the voxel index
	uint3 id;

	//Initialising <id>
	id.x = blockIdx.x * blockDim.x + threadIdx.x;
	id.y = blockIdx.y * blockDim.y + threadIdx.y;
	id.z = blockIdx.z * blockDim.z + threadIdx.z;

	//There are xdim-1 * ydim -1 * zdim-1 voxel in a voxumteric data of size
	//[xdim,ydim,zdim]
	if (id.x < (xdim - 1) || id.y < (ydim - 1) || id.z << (zdim - 1)) {

		//Variables
		float4 voxels[8];
		float3 pos[12];
		int index[8];
		unsigned int cubeIndex = 0;
		float3 vertices[15];
		int numTriangles = 0;
		int numVertices = 0;

		//Getting the index of 8 vertices
		index[0] = xdim * (ydim * id.z + id.y) + id.x;
		index[1] = xdim * (ydim * id.z + id.y + 1) + id.x;
		index[2] = xdim * (ydim * id.z + id.y + 1) + id.x + 1;
		index[3] = xdim * (ydim * id.z + id.y) + id.x + 1;
		index[4] = xdim * (ydim * (id.z + 1) + id.y) + id.x;
		index[5] = xdim * (ydim * (id.z + 1) + id.y + 1) + id.x;
		index[6] = xdim * (ydim * (id.z + 1) + id.y + 1) + id.x + 1;
		index[7] = xdim * (ydim * (id.z + 1) + id.y) + id.x + 1;


		for (int i = 0; i < 8; ++i)
		{
			//Getting data of the vertices in this voxel
			voxels[i].x = device[index[i]].x;
			voxels[i].y = device[index[i]].y;
			voxels[i].z = device[index[i]].z;
			voxels[i].w = device[index[i]].w;

			//Comparing the grid value at 8 points
			if (voxels[i].w >= isovalue) {
				//cubeIndex is being left shifted based on the vertex
				cubeIndex |= (1 << i);
			}
		}


		//Getting edges
		unsigned int edges = edgeTable[cubeIndex];

		//Comparing edges with 12 bit by and operation and position coordinate
		if (edges == 0) {
			return;
		}
		if (edges & 1) {
			pos[0] = linearInterpolation(isovalue, voxels[0], voxels[1]);
		}
		if (edges & 2) {
			pos[1] = linearInterpolation(isovalue, voxels[1], voxels[2]);
		}
		if (edges & 4) {
			pos[2] = linearInterpolation(isovalue, voxels[2], voxels[3]);
		}
		if (edges & 8) {
			pos[3] = linearInterpolation(isovalue, voxels[3], voxels[0]);
		}
		if (edges & 16) {
			pos[4] = linearInterpolation(isovalue, voxels[4], voxels[5]);
		}
		if (edges & 32) {
			pos[5] = linearInterpolation(isovalue, voxels[5], voxels[6]);
		}
		if (edges & 64) {
			pos[6] = linearInterpolation(isovalue, voxels[6], voxels[7]);
		}
		if (edges & 128) {
			pos[7] = linearInterpolation(isovalue, voxels[7], voxels[4]);
		}
		if (edges & 256) {
			pos[8] = linearInterpolation(isovalue, voxels[0], voxels[4]);
		}
		if (edges & 512) {
			pos[9] = linearInterpolation(isovalue, voxels[1], voxels[5]);
		}
		if (edges & 1024) {
			pos[10] = linearInterpolation(isovalue, voxels[2], voxels[6]);
		}
		if (edges & 2048) {
			pos[11] = linearInterpolation(isovalue, voxels[3], voxels[7]);
		}

		for (int n = 0; n < 15; n += 3)
		{
			int edgeNumber = triTable[cubeIndex][n];
			if (edgeNumber < 0)
				break;

			vertices[numVertices++] = pos[edgeNumber];
			vertices[numVertices++] = pos[triTable[cubeIndex][n + 1]];
			vertices[numVertices++] = pos[triTable[cubeIndex][n + 2]];
			++numTriangles;
		}

		//Getting the number of triangles
		triangle[index[0]] = numTriangles;

		//Vertex List
		for (int n = 0; n < numVertices; ++n) {
			vertex[MAX_VERTEX * index[0] + n] = vertices[n];
		}
	}
	return;
}

void writeOutputList(const int totalSize, std::string IndexFile, std::string
	VertexFile)
{
	//Creating Output Streams
	std::fstream Output[4];

	//Opening Files
	Output[0].open(IndexFile.c_str(), std::ios::out);
	Output[1].open(VertexFile.c_str()
		+ std::to_string(1) + ".dat", std::ios::out);

	Output[2].open(VertexFile.c_str()
		+ std::to_string(2) + ".dat", std::ios::out);

	Output[3].open(VertexFile.c_str()
		+ std::to_string(3) + ".dat", std::ios::out);

	//Printing Counter to file 
	if (Output[0].is_open()) {
		StatusLog("Writting Index File");
		for (int i = 0; i < totalSize; i++) {
			Output[0] << counter[i] << "\t";
		}
	}
	else
	{
		StatusLog("Error: Failed to open output files:" + IndexFile);
		exit(-1);
	}

	delete[]counter;

	Output[0].close();

	//Printing Verxtex list
	if (Output[1].is_open() && Output[2].is_open() && Output[3].is_open()) 
	{
		StatusLog("Writting vertex files");
		for (int i = 0; i < totalSize; i++) {
			Output[1] << bin[i * MAX_VERTICES].x << "\t";
			Output[2] << bin[i * MAX_VERTICES].y << "\t";
			Output[3] << bin[i * MAX_VERTICES].z << "\t";
		}
	}
	else if(Output[1].is_open() != 1)
	{
		StatusLog("Error: Failed to open " + VertexFile + std::to_string(1));
		exit(-1);
	}
	else if(Output[2].is_open() != 1)
	{
		StatusLog("Error: Failed to open " + VertexFile + std::to_string(2));
		exit(-1);
	}
	else
	{
		StatusLog("Error: Failed to open " + VertexFile + std::to_string(3));
		exit(-1);
	}
	
	delete[]bin;

	//Closing streams
	Output[1].close();
	Output[2].close();
	Output[3].close();

	return;
}

void marchingCubes(thrust::host_vector<float4>& host_vec,inputParameters data){

	//Size of the grid
	int totalSize = int(data.x_dim * data.y_dim * data.z_dim);

	//4 x 4 x 4 block dimension
	dim3 blockSize(4, 4, 4);

	//Initialising grid dimension
	dim3 gridSize((data.x_dim + blockSize.x - 1) / blockSize.x,
		(data.y_dim + blockSize.y - 1) / blockSize.y,
		(data.z_dim + blockSize.z - 1) / blockSize.z);

	//Updating log file 
	StatusLog("Invoking Kernel");
	
	//Updating Memory Status
	GetMemoryStatus();

	//Calling cuda kernel hipEvent_t start, stop;
	hipEvent_t start, stop;
	float elapsedTime;

	//Creating Start and Stop events
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);

	// Host to Device Memory Copy
	thrust::device_vector<float4> device_vec = host_vec;
	
	//Casting a raw pointer for the device vector
	float4 *device_pointer = thrust::raw_pointer_cast(device_vec.data());

	//Output in the GPU will stored in vertex and triangle
	float3* vertex;
	int* triangle;
		
	hipMalloc(&vertex, sizeof(float3) * totalSize * MAX_VERTICES);
	hipMemset(vertex, 0, sizeof(float3) * totalSize * MAX_VERTICES);
	hipMalloc(&triangle, sizeof(int) * totalSize);
	hipMemset(triangle, 0, sizeof(int) * totalSize);
	

	//Invoking Kernel
	kernel <<<gridSize, blockSize >>> (data.isovalue, data.x_dim, data.y_dim,
		data.z_dim, device_pointer, vertex, triangle);

	
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		std::cout << hipGetErrorString(error) << std::endl;
		exit(-1);
	}
	
	//Memcopy - Device to Host
	bin = (float3*)malloc(sizeof(float3) * totalSize * MAX_VERTICES);
	hipMemcpy(vertex, bin, sizeof(float3) * totalSize * MAX_VERTICES, 
													   hipMemcpyDeviceToHost);

	counter = (int*)malloc(sizeof(int) * totalSize);
	hipMemcpy(counter, counter, sizeof(int) * totalSize,
													   hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&elapsedTime, start, stop);
	
	StatusLog("Returned to host");
	StatusLog("Elapsed Time: " + std::to_string(elapsedTime));
	
	//Destroying events
	hipEventDestroy(start);
	hipEventDestroy(stop);

	//Clearing the Host Memory
	try {
		host_vec.clear();
		host_vec.shrink_to_fit();
	}
	catch (thrust::system_error e) {
		StatusLog(e.what());
		exit(-1);
	}
	
	GetMemoryStatus();

	//Clearing Device Memory
	hipFree(triangle);
	hipFree(vertex);
	hipError_t MemoryError = hipGetLastError();
	
	if (hipSuccess != MemoryError)
	{
		StatusLog(hipGetErrorString(MemoryError));
		exit(-1);
	}
	
	try {
		device_vec.clear();
		device_vec.shrink_to_fit();
	}
	catch (thrust::system_error e)
	{
		StatusLog(e.what());
		exit(-1);
	}
	
	GetMemoryStatus();
	
	//Invoking Function to Create and Print Index List and Vertex List
	StatusLog("Creating Lists");

	//Writting Output
	writeOutputList(totalSize, data.OutputIndexFile, data.OutputVertexFile);
	
	return;
}

//-----------------------------------------------------------------------------
// Main function
//-----------------------------------------------------------------------------

int main(int argc, char** argv)
{
	
	//Declaring and defining an object to struct <inputParameters>
	inputParameters data = input_parameter_initialization(std::string(argv[1]));

	thrust::host_vector<float4> host_vec;

	//Declaring and intialising the volumetric data
	inputRead(data, host_vec);

	//Calling marchingCubes to invoke kernel
	marchingCubes(host_vec, data);

	//Clearing Device Memory

	StatusLog("Execution Successfully");
	//Execution Successful

	return (0);
}
